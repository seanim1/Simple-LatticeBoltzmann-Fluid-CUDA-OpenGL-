#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>  // Helper functions for CUDA Error handling
#include <hip/hip_cooperative_groups.h>
#include <helper_string.h>

#include "RenderToTexture_CUDA+GL.h"

// Texture object for reading image
hipTextureObject_t texObject;
static hipArray *array = NULL;
const bool DEBUG = false;
const float tau = 0.6f;
float* d_Ce; // 0: Center
float* d_No; // 1: North
float* d_NE;
float* d_Ea;
float* d_SE;
float* d_So;
float* d_SW;
float* d_We;
float* d_NW;
bool* d_cylinder;
float* d_rho;
float* d_ux;
float* d_uy;

__global__ void DriftNorth(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (blockIdx.x == blockDim.x - 1) {
        data[elem] = data[threadIdx.x];
    }
    else {
        float temp = data[elem + width];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(data[elem] * 128, 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("No: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftNorthEast(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (threadIdx.x == (width - 1)) {
        // boundary absorption
    }
    else if (blockIdx.x == blockDim.x - 1 && threadIdx.x == 0) {
        data[elem] = data[width - 1];
    }
    else if (blockIdx.x == blockDim.x - 1) {
        data[elem] = data[threadIdx.x];
    } 
    else if (threadIdx.x == 0) {
        data[elem] = data[(blockIdx.x + 1) * width - 1];
    }
    else {
        float temp = data[elem + width - 1];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(powf(data[elem] * 2, 8.f), 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("NE: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftEast(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (threadIdx.x == (width - 1)) {
        // boundary absorption
    }
    else if (threadIdx.x == 0) {
        data[elem] = data[(blockIdx.x + 1) * width - 1];
    } // East Boundary: Velocity is absorbed
    else {
        float temp = data[elem - 1];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(data[elem] * 128, 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("Ea: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftSouthEast(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (threadIdx.x == (width - 1)) {
        // boundary absorption
    }
    else if (blockIdx.x == 0 && threadIdx.x == 0) {
        data[elem] = data[blockDim.x * width - 1];
    }
    else if (blockIdx.x == 0) {
        data[elem] = data[(blockDim.x - 1) * width + threadIdx.x];
    }
    else if (threadIdx.x == 0) {
        data[elem] = data[(blockIdx.x + 1) * width - 1];
    }
    else {
        float temp = data[elem - width - 1];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(powf(data[elem] * 2, 8.f), 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("SE: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftSouth(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (blockIdx.x == 0) {
        data[elem] = data[(blockDim.x - 1) * width + threadIdx.x];
        //printf("Blk(% d, % d) Tred(% d, % d): %3.2f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, data[elem]);
    }
    else {
        float temp = data[elem - width];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(powf(data[elem] * 2, 8.f), 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("So: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftSouthWest(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (threadIdx.x == 0) {
        // boundary absorption
    }
    else if (blockIdx.x == 0 && threadIdx.x == (width - 1)) {
        data[elem] = data[(blockDim.x - 1) * width];
    }
    else if (blockIdx.x == 0) {
        data[elem] = data[(blockDim.x - 1) * width + threadIdx.x];
    }
    else if (threadIdx.x == (width - 1)) {
        data[elem] = data[blockIdx.x * width];
    }
    else {
        float temp = data[elem - width + 1];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(powf(data[elem] * 2, 8.f), 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("SW: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftWest(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (threadIdx.x == 0) {
        // boundary absorption
    }
    else if (threadIdx.x == (width - 1)) {
        data[elem] = data[blockIdx.x * width];
    }
    else {
        float temp = data[elem + 1];
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(data[elem] * 128, 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("We: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void DriftNorthWest(Pixel* pixels, int width, int height, float* data) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    if (threadIdx.x == 0) {
        // boundary absorption
    }
    else if (blockIdx.x == (blockDim.x - 1) && threadIdx.x == (width - 1)) {
        data[elem] = data[0];
    } else if (blockIdx.x == (blockDim.x - 1)) {
        data[elem] = data[threadIdx.x];
    } else if (threadIdx.x == (width - 1)) {
        data[elem] = data[blockIdx.x * width];
    }
    else {
        float temp = data[elem + width + 1];
        //printf("Blk(% d, % d) Tred(% d, % d) Src: %2.2f, Dst: %2.2f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, d_No[elem], temp);
        data[elem] = temp;
    }
    //pData[threadIdx.x] = (unsigned char)min(max(powf(data[elem] * 2, 8.f), 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("NW: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, data[elem]);
        }
    }
}
__global__ void Momentum(Pixel* pixels, int width, int height, float* d_rho, float* d_ux, float* d_uy, bool* d_cylinder, float* d_Ce, float* d_No, float* d_NE, float* d_Ea, float* d_SE, float* d_So, float* d_SW, float* d_We, float* d_NW) {
    //unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    //printf("%3.3f \n", d_ux[elem]);
    d_rho[elem] = (d_Ce[elem] + d_No[elem] + d_NE[elem] + d_Ea[elem] + d_SE[elem] + d_So[elem] + d_SW[elem] + d_We[elem] + d_NW[elem]);
    d_ux[elem] = (d_NE[elem] + d_Ea[elem] + d_SE[elem] - d_SW[elem] - d_We[elem] - d_NW[elem]) / d_rho[elem];
    d_uy[elem] = (d_No[elem] + d_NE[elem] - d_SE[elem] - d_So[elem] - d_SW[elem] + d_NW[elem]) / d_rho[elem];
    //pData[threadIdx.x] = (unsigned char)min(max((rho / 9.0f * 256.0f), 0.f), 255.f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("Rho: Blk(% d, % d) Tred(% d, % d) rho:%3.5f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, d_rho[elem]);
        }
    }
}
/*
* 8 -- 1 -- 2
* -    -    -
* 7 -- 0 -- 3
* -    -    -
* 6 -- 5 -- 4
*/
__global__ void ApplyBoundary(Pixel* pixels, int width, int height, float* d_ux, float* d_uy, bool* d_cylinder, float* d_No, float* d_NE, float* d_Ea, float* d_SE, float* d_So, float* d_SW, float* d_We, float* d_NW) {
    int elem = blockIdx.x * width + threadIdx.x;
    //printf("%3.3f \n", d_ux[elem]);
    if (d_cylinder[elem]) {
        // Reflect
        // Swap 1:North 5:South
        float temp = d_No[elem];
        d_No[elem] = d_So[elem];
        d_So[elem] = temp;
        // Swap 2:NorthEast 6:SouthWest
        temp = d_NE[elem];
        d_NE[elem] = d_SW[elem];
        d_SW[elem] = temp;
        // Swap 3:East 7:West
        temp = d_Ea[elem];
        d_Ea[elem] = d_We[elem];
        d_We[elem] = temp;
        // Swap 4:SouthEast 8:NorthWest
        temp = d_SE[elem];
        d_SE[elem] = d_NW[elem];
        d_NW[elem] = temp;
        // there is no fluid movement within the boundary
        d_ux[elem] = 0;
        d_uy[elem] = 0;
    }
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("BoundUX: Blk(% d, % d) Tred(% d, % d) ux:%3.5f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, d_ux[elem]);
            printf("BoundUY: Blk(% d, % d) Tred(% d, % d) uy:%3.5f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, d_uy[elem]);
        }
    }
}
__device__ float F_equilibrium_timeStep(int dir_x, int dir_y, float rho, float weight, float ux, float uy, float lattice, float tau) {
    float feq = rho * weight * (1.f + 3.f * (dir_x * ux + dir_y * uy) + 9.f * powf(dir_x * ux + dir_y * uy, 2.f) / 2.0f - 3.f * (powf(ux, 2.f) + powf(uy, 2.f)) / 2.0f);
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("feq: Blk(% d, % d) Tred(% d, % d) feq:%3.5f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, feq);
        }
    }
    return -(1.0f / tau) * (lattice - feq);
}
__global__ void SolveTimeStep(Pixel* pixels, int width, int height, float tau, float* d_rho, float* d_ux, float* d_uy, float* d_Ce, float* d_No, float* d_NE, float* d_Ea, float* d_SE, float* d_So, float* d_SW, float* d_We, float* d_NW) {
    int elem = blockIdx.x * width + threadIdx.x;
    d_Ce[elem] += F_equilibrium_timeStep( 0, 0, d_rho[elem], 4 / 9.0f, d_ux[elem], d_uy[elem], d_Ce[elem], tau);
    d_No[elem] += F_equilibrium_timeStep( 0, 1, d_rho[elem], 1 / 9.0f, d_ux[elem], d_uy[elem], d_No[elem], tau);
    d_NE[elem] += F_equilibrium_timeStep( 1, 1, d_rho[elem], 1 / 36.f, d_ux[elem], d_uy[elem], d_NE[elem], tau);
    d_Ea[elem] += F_equilibrium_timeStep( 1, 0, d_rho[elem], 1 / 9.0f, d_ux[elem], d_uy[elem], d_Ea[elem], tau);
    d_SE[elem] += F_equilibrium_timeStep( 1,-1, d_rho[elem], 1 / 36.f, d_ux[elem], d_uy[elem], d_SE[elem], tau);
    d_So[elem] += F_equilibrium_timeStep( 0,-1, d_rho[elem], 1 / 9.0f, d_ux[elem], d_uy[elem], d_So[elem], tau);
    d_SW[elem] += F_equilibrium_timeStep(-1,-1, d_rho[elem], 1 / 36.f, d_ux[elem], d_uy[elem], d_SW[elem], tau);
    d_We[elem] += F_equilibrium_timeStep(-1, 0, d_rho[elem], 1 / 9.0f, d_ux[elem], d_uy[elem], d_We[elem], tau);
    d_NW[elem] += F_equilibrium_timeStep(-1, 1, d_rho[elem], 1 / 36.f, d_ux[elem], d_uy[elem], d_NW[elem], tau);
    /*if (blockIdx.x == 1 && threadIdx.x == 1) {
        printf("Blk(% d, % d) Tred(% d, % d) d_NO:%3.2f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, d_No[elem]);
    }*/
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("d_Ce: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_Ce[elem]);
            printf("d_No: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_No[elem]);
            printf("d_NE: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_NE[elem]);
            printf("d_Ea: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_Ea[elem]);
            printf("d_SE: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_SE[elem]);
            printf("d_So: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_So[elem]);
            printf("d_SW: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_SW[elem]);
            printf("d_We: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_We[elem]);
            printf("d_NW: Blk(%d) Tred(%d): %2.5f\n", blockIdx.x, threadIdx.x, d_NW[elem]);
        }
    }
}
__global__ void DisplayVelocity(Pixel* pixels, int width, float* d_ux, float* d_uy, float brightness) {
    unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    float velocity = (powf(d_ux[elem], 2.f) + powf(d_uy[elem], 2.f));
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("Display Vel: Blk(% d, % d) Tred(% d, % d) vel:%3.5f \n\n\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, velocity);
        }
    }
    velocity = (-1 * expm1f(-velocity * brightness * 0.1f)) * 255.f; // exposure tonemapping
    if (DEBUG) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("Tonemap Vel: Blk(% d, % d) Tred(% d, % d) vel:%3.5f \n\n\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, velocity);
        }
    }
    //printf("Blk(% d, % d) Tred(% d, % d): %3.2f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, velocity);
    //printf("Blk(% d, % d) Tred(% d, % d): %3.2f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, d_uy[elem]);
    pData[threadIdx.x] = (unsigned char) min( max(velocity, 0.f), 255.f );
}
/*
__global__ void DisplayVorticity(Pixel* pixels, int width, float* d_ux, float* d_uy, float brightness) {
    unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    int elem = blockIdx.x * width + threadIdx.x;
    float ux1 = 0.f; //ignore the top two rows, ignore the first and the last columns
    if (blockIdx.x >= 2 && threadIdx.x == 0 && threadIdx.x == (width - 1)) {
        ux1 = d_ux[elem];
    }
    float ux2 = 0.f; // ignore the bottom two rows, ignore the first and the last columns
    if (blockIdx.x < (blockDim.x - 2) && threadIdx.x == 0 && threadIdx.x == (width - 1)) {
        break;
    }
    float curl = 0.f;
    pData[threadIdx.x] = (unsigned char)min(max(curl, 0.f), 255.f);
}*/
__global__ void Kernel_01(Pixel *pixels, int width, int height, float fScale, hipTextureObject_t texObj) {
    unsigned char* pData = (unsigned char*)(((char*)pixels) + blockIdx.x * width);
    pData[threadIdx.x] = min(max((tex2D<unsigned char>(texObj, (float)threadIdx.x, (float)blockIdx.x) * fScale), 0.f), 255.f);
}
// Wrapper for the __global__ call that sets up the texture and threads
extern "C" void Kernal(Pixel *odata, int width, int height, enum DisplayMode mode, float fScale) {
    dim3 THREADS(width, 1, 1); // Z-dimension is 1 by default.
    dim3 BLOCKS(height, 1, 1);
    //Kernel_01 << <BLOCKS, THREADS >> > (odata, width, height, fScale, texObject);
    /*const int ARRAY_SIZE = width * height;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    float* tempNo = (float*)malloc(ARRAY_BYTES);*/
    DriftNorth      << < BLOCKS, THREADS >> > (odata, width, height, d_No);
    DriftNorthEast << < BLOCKS, THREADS >> > (odata, width, height, d_NE);
    DriftEast       << < BLOCKS, THREADS >> > (odata, width, height, d_Ea);
    DriftSouthEast  << < BLOCKS, THREADS >> > (odata, width, height, d_SE);
    DriftSouth      << < BLOCKS, THREADS >> > (odata, width, height, d_So);
    DriftSouthWest << < BLOCKS, THREADS >> > (odata, width, height, d_SW);
    DriftWest       << < BLOCKS, THREADS >> > (odata, width, height, d_We);
    DriftNorthWest << < BLOCKS, THREADS >> > (odata, width, height, d_NW);
    Momentum << < BLOCKS, THREADS >> > (odata, width, height, d_rho, d_ux, d_uy, d_cylinder, d_Ce, d_No, d_NE, d_Ea, d_SE, d_So, d_SW, d_We, d_NW);
    ApplyBoundary << < BLOCKS, THREADS >> > (odata, width, height, d_ux, d_uy, d_cylinder, d_No, d_NE, d_Ea, d_SE, d_So, d_SW, d_We, d_NW);
    SolveTimeStep << < BLOCKS, THREADS >> > (odata, width, height, tau, d_rho, d_ux, d_uy, d_Ce, d_No, d_NE, d_Ea, d_SE, d_So, d_SW, d_We, d_NW);
    DisplayVelocity << < BLOCKS, THREADS >> > (odata, width, d_ux, d_uy, fScale);
    //DisplayVorticity << < BLOCKS, THREADS >> > (odata, width, d_ux, d_uy, fScale);
    
    //std::cout << "Pixel: " << mode << std::endl;
    switch (mode) {
        case VELOCITY:
            break;

        case VORTICITY:
            break;
    }
}

float distanceSq(float offset_x, float offset_y, float x, float y) {
    return std::pow(x - offset_x, 2) + std::pow(y - offset_y, 2);
}

extern "C" void initMemory(int wWidth, int wHeight) {
    // initialize lattice velocities on the host and transfer it to device
    const int ARRAY_SIZE = wWidth * wHeight;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    float* Ce = (float*)malloc(ARRAY_BYTES);
    float* No = (float*)malloc(ARRAY_BYTES);
    float* NE = (float*)malloc(ARRAY_BYTES);
    float* Ea = (float*)malloc(ARRAY_BYTES);
    float* SE = (float*)malloc(ARRAY_BYTES);
    float* So = (float*)malloc(ARRAY_BYTES);
    float* SW = (float*)malloc(ARRAY_BYTES);
    float* We = (float*)malloc(ARRAY_BYTES);
    float* NW = (float*)malloc(ARRAY_BYTES);
    bool* cylinder = (bool*)malloc(ARRAY_SIZE);
    float* rho = (float*)malloc(ARRAY_BYTES); // density 
    float* ux = (float*)malloc(ARRAY_BYTES);
    float* uy = (float*)malloc(ARRAY_BYTES);
    //float rho = 11.3f; // sum of all lattice velocities
    for (int i = 0; i < ARRAY_SIZE; i++) {
        Ce[i] = 0.95f + rand() % 100 / 1000.f;
        No[i] = 0.95f + rand() % 100 / 1000.f;
        NE[i] = 0.95f + rand() % 100 / 1000.f;
        Ea[i] = 0.95f + rand() % 100 / 1000.f + 2.3f; // velocity of the right cell.
        SE[i] = 0.95f + rand() % 100 / 1000.f;
        So[i] = 0.95f + rand() % 100 / 1000.f;
        SW[i] = 0.95f + rand() % 100 / 1000.f;
        We[i] = 0.95f + rand() % 100 / 1000.f;
        NW[i] = 0.95f + rand() % 100 / 1000.f;
        if (distanceSq(wWidth / 4, wHeight / 2, i % wWidth, i / wWidth ) < 256) {
            cylinder[i] = true;
            if (DEBUG) {
                std::cout << "Cylinder: " << i / wWidth << ", " << i % wWidth << std::endl;
            }
        }
        else {
            cylinder[i] = false;
        }
        rho[i] = 11.3f;
        ux[i] = 0.f;
        uy[i] = 0.f;
    }
    //
    hipMalloc((void**)&d_Ce, ARRAY_BYTES);
    hipMalloc((void**)&d_No, ARRAY_BYTES);
    hipMalloc((void**)&d_NE, ARRAY_BYTES);
    hipMalloc((void**)&d_Ea, ARRAY_BYTES);
    hipMalloc((void**)&d_SE, ARRAY_BYTES);
    hipMalloc((void**)&d_So, ARRAY_BYTES);
    hipMalloc((void**)&d_SW, ARRAY_BYTES);
    hipMalloc((void**)&d_We, ARRAY_BYTES);
    hipMalloc((void**)&d_NW, ARRAY_BYTES);
    hipMalloc((void**)&d_cylinder, ARRAY_SIZE);
    hipMalloc((void**)&d_rho, ARRAY_BYTES);
    hipMalloc((void**)&d_ux, ARRAY_BYTES);
    hipMalloc((void**)&d_uy, ARRAY_BYTES);
    hipMemcpy(d_Ce, Ce, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_No, No, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_NE, NE, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_Ea, Ea, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_SE, SE, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_So, So, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_SW, SW, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_We, We, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_NW, NW, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_cylinder, cylinder, ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_rho, rho, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_ux, ux, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_uy, uy, ARRAY_BYTES, hipMemcpyHostToDevice);
    free(Ce);
    free(No);
    free(NE);
    free(Ea);
    free(SE);
    free(So);
    free(SW);
    free(We);
    free(NW);
    free(cylinder);
    free(ux);
    free(uy);
}

extern "C" void setupTexture(int iw, int ih, Pixel * data, int Bpp) {
    hipChannelFormatDesc desc;

    if (Bpp == 1) {
        desc = hipCreateChannelDesc<unsigned char>();
    }
    else {
        desc = hipCreateChannelDesc<uchar4>();
    }

    checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
    checkCudaErrors(hipMemcpy2DToArray(array, 0, 0, data, iw * Bpp * sizeof(Pixel), iw * Bpp * sizeof(Pixel), ih, hipMemcpyHostToDevice));
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = array;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));
}

extern "C" void deleteTexture(void) {
    checkCudaErrors(hipFreeArray(array));
    checkCudaErrors(hipDestroyTextureObject(texObject));
}
